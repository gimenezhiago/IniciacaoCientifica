#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Função para obter o tempo atual
double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec / 1e6;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Uso: %s <tamanho_matriz>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    size_t size = N * N * sizeof(double);

    // Alocação e inicialização no host
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    srand(1);
    for (int i = 0; i < N * N; i++) {
        h_A[i] = h_B[i] = rand() % 100;
        h_C[i] = 0.0;
    }

    // Alocação na GPU
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copiar para a GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Criação do handle cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Parâmetros de alpha e beta
    double alpha = 1.0, beta = 0.0;

    // Execução com cuBLAS
    double start = get_time();
    // cublasDgemm(handle, transA, transB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    hipDeviceSynchronize();
    double end = get_time();

    // Copiar o resultado de volta para o host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("cuBLAS: Tempo = %.4f s\n", end - start);

    // Liberar memória
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}
