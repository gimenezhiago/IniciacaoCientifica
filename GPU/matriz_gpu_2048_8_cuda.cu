#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define N 2048

__global__ void matrixMulCUDA(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec / 1e6;
}

int main() {
    size_t size = N * N * sizeof(double);
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);
    
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }
    
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    dim3 blockSize(8, 8);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    
    double start = get_time();
    matrixMulCUDA<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    double end = get_time();
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    printf("Tem %d\n", gridSize.x);
    printf("Matriz de %d x %d\n", N, N);
    printf("CUDA: Tempo = %.4f s\n", end - start);
    
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    return 0;
}
